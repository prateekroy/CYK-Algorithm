#include "hip/hip_runtime.h"
#include "cyk.h"
#include <map>
#include <set>
// #include <cilk/cilk.h>
using namespace std;

typedef std::vector<string> SymbolsSet;
typedef std::map<string, RuleVector> RulesMap;
typedef std::map<string, int> SymbolIndices;
typedef std::map<string, RuleVector> LexiconsMap;

RulesMap rulesMap;
LexiconsMap lexiconsMap;
SymbolsSet symbols;
SymbolIndices symIndices;

struct IntRule{
	int leftsymIndex;
	int right1symIndex;
	int right2symIndex;
	int score;
};


void unaryRelax (int *** scores, int begin, int end, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp){
	cout<<endl<<endl<<endl<<"In Unary Relax"<<endl;
	int prob=0;
	RuleVector rulesList;
	std::vector<string>::iterator itr;
	
	//for all symbols
	for (itr = symbols.begin(); itr != symbols.end(); ++itr){
		rulesList = rulesMap[*itr];
		//for all rules of that symbol
		for(int k=0;k<rulesList.size();k++){
			//get only unary rules
			if(rulesList[k]->is_first_order()){

				//The rule is of the form A-> B. get the score of A>B and that symbol B is present at score [begin][end]
				prob = rulesList[k]->score + scores[begin][end][symIndices[rulesList[k]->right1]];

				// if the above score is greater than the score of B, then add A to the location [begin][end]
		     	if(prob > scores[begin][end][symIndices[rulesList[k]->left]]){
		     		scores[begin][end][symIndices[rulesList[k]->left]] = prob;
		     		cout << rulesList[k]->left << " ----> "<<rulesList[k]->right1 << endl;
		     		bp[begin][end][symIndices[rulesList[k]->left]]->setBP(rulesList[k]->right1,"",0);
		     	}
         	}
		}
	}
	cout<<"OUt of unary relax"<<endl;
}


void binaryRelax(int *** scores, int nWords, int length, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp){
	cout<<endl<<endl<<endl<<"In Binary Relax"<<endl;
	int end=0;
	int max = -1;
	RuleVector rulesList;
	int lscore=0, rscore =0,score=0;
	std::vector<string> :: iterator itr;
	string right1="", right2="";
	int bpSplit=-1;

	for(int start =0; start <=nWords-length; start++){
		
		end = start + length;
		
		//for all symbols
		for (itr = symbols.begin(); itr != symbols.end(); ++itr){
			max = 0;
			rulesList = rulesMap[*itr];
			for(int j=0;j<rulesList.size();j++){

				//TODO check if for unary rules, the right child is in right1
				if(rulesList[j]->is_second_order()){
					for (int split =start+1;split<=end-1;split++){
						lscore = scores[start][split][symIndices[rulesList[j]->right1]];
						rscore = scores[split][end][symIndices[rulesList[j]->right2]];
						score = rulesList[j]->score + lscore + rscore;

						if(score > max){
							max = score;
							bpSplit = split;
							right1 = rulesList[j]->right1;
							right2 = rulesList[j]->right2;
							cout<<rulesList[j]->left<<" ------> "<<rulesList[j]->right1<<" "<<rulesList[j]->right2;
						}
					}
				}
			}
			scores[start][end][symIndices[*itr]]=max;
			bp[start][end][symIndices[*itr]]->setBP(right1,right2,bpSplit);

			// cout << right1 << " " << right2 << endl;
			
		}

		unaryRelax(scores, start, end, rules,symbols, bp);
	}
}



IntRule* ConvertRule(RuleVector& rules)
{
	IntRule* ruleArr = new IntRule[rules.size()];
	for (int i = 0; i < rules.size(); ++i)
	{
		ruleArr[i].leftsymIndex = symIndices[rules[i]->left];
		ruleArr[i].right1symIndex = symIndices[rules[i]->right1];
		ruleArr[i].right2symIndex = symIndices[rules[i]->right2];
		ruleArr[i].score = rules[i]->score;
	}

	return ruleArr;
}

int* ConvertScore(int*** score, int A, int B, int C)
{
  int* hscore;// = new int[A*B*C*10];
  hipHostMalloc((void**)&hscore, A*B*C*sizeof(int));
  for (int i = 0; i < A; ++i)
  {
    for (int j = 0; j < B; ++j)
    {
      for (int k = 0; k < C; ++k)
      {
        hscore[i + B * (j + C * k)] = score[i][j][k];
      }
    }
  }


  return hscore;
}

int* ConvertToCudaDevice(int* a, int n){
  int* d;
  hipMalloc((void**)&d, n*sizeof(int));
  hipMemcpy(d, a, n*sizeof(int), hipMemcpyHostToDevice);

  return d;
}

IntRule* ConvertRuleTOCudaDevice(IntRule* rules, int N){

  IntRule* drules;
  hipMalloc((void**)&drules, N*sizeof(IntRule));
  hipMemcpy(drules, rules, N*sizeof(IntRule), hipMemcpyHostToDevice);	

  return drules;
}

void convert3DArrayToCudaDevice(int* scores,int* newScores, int A, int B , int C){
	hipMalloc((void**)&newScores, A*B*C*sizeof(int));
  	hipMemcpy(newScores, scores, A*B*C*sizeof(int), hipMemcpyHostToDevice);

}








__global__
static void MyFunc2D(IntRule* drules, int r, int* dscores, int A, int B, int C, int start, int end, int* sh_max) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  	
  //Debug Code
  if (i == 0)
  {
	// for (int i = 0; i < r; ++i)
	// {
	// 	printf("%d -> %d %d\n", drules[i].leftsymIndex, drules[i].right1symIndex, drules[i].right2symIndex);
	// }
	// for (int u = 0; u < A; ++u)
	// {
	// 	for (int v = 0; v < B; ++v)
	// 	{
	// 	  for (int w = 0; w < C; ++w)
	// 	  {
	// 	    printf("%d ", dscores[u + B * (v + C * w)]);
	// 	  }
	// 	  printf("\n");
	// 	}
	// 	printf("\n");
	// }

  }



  if (i < r)
  {
  		  // printf("Index %d ", i);
		{
			int l_sym = drules[i].right1symIndex;
			int r_sym = drules[i].right2symIndex;
			int symbol = drules[i].leftsymIndex;

			int local_max = 0;
			// int bpSplit=-1;
			// string right1 = "";
			// string right2 = "";

			for (int split =start+1;split<=end-1;split++){
				int lscore = dscores[start + B * (split + C * l_sym)];
				int rscore = dscores[split + B * (end + C * r_sym)];
				int score = drules[i].score + lscore + rscore;

				if(score > local_max){
					local_max = score;
					printf("symbol : %d local_max : %d l_score : %d r_score : %d score : %d\n", symbol, local_max, lscore, rscore, score);

					//needed for backpointer
					// bpSplit = split;
					// right1 = l_sym;
					// right2 = r_sym;

				}
			}	

			//atomic max // for now use lock

			atomicMax(&sh_max[symbol], local_max);
			// bp[start][end][symIndices[symbol]]->setBP(right1,right2,bpSplit);
		}		
  }
}




void threadBasedRuleBR(int *** scores, int nWords, int length, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp, int A, int B, int C){



	IntRule* hrules = ConvertRule(rules);
	IntRule* drules = ConvertRuleTOCudaDevice(hrules, rules.size());


	for(int start =0; start <=nWords-length; start++){
		// for (int i = 0; i < A; ++i)
		// {
		// 	for (int j = 0; j < B; ++j)
		// 	{
		// 		for (int k = 0; k < C; ++k)
		// 		{
		// 			cout << scores[i][j][k] << " ";
		// 		}
		// 	}
		// }

		// cout << endl;
		// cout << "\n******************************************\n";


		int* arr = ConvertScore(scores, A, B, C);
		int* dscores = ConvertToCudaDevice(arr, A*B*C);
		// int* dscores;
		// hipMalloc((void**)&dscores, A*B*C*sizeof(int));
		// hipMemset(dscores, 0, A*B*C*sizeof(int));
		// hipMemcpy(dscores, arr, A*B*C*sizeof(int), hipMemcpyHostToDevice);

		
		int end = start + length;
		int* shared_max = new int[symbols.size()];
		for (int i = 0; i < symbols.size(); ++i)
		{
			shared_max[i] = 0;
		}

		int* sh_max = ConvertToCudaDevice(shared_max, symbols.size());

	  	MyFunc2D<<<1, rules.size()*2>>>(drules, rules.size(), dscores, A, B, C, start, end, sh_max);


	  	hipMemcpy(shared_max, sh_max, symbols.size()*sizeof(int), hipMemcpyDeviceToHost);

	  	for (int i = 0; i < symbols.size(); ++i)
	  	{
	  		cout << shared_max[i] << " ";
	  	}
	  	cout << "\n@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n";

		//make this parallel for
		for (std::vector<string>::iterator itr = symbols.begin(); itr != symbols.end(); ++itr){
			scores[start][end][symIndices[*itr]] = max(scores[start][end][symIndices[*itr]], shared_max[symIndices[*itr]]);	
		}



		unaryRelax(scores, start, end, rules,symbols, bp);
	}	
}


#include <mutex>
std::mutex guard;
void threadBasedRuleBRcpu(int *** scores, int nWords, int length, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp){




	for(int start =0; start <=nWords-length; start++){
		
		int end = start + length;
		int* shared_max = new int[symbols.size()];
		for (int i = 0; i < symbols.size(); ++i)
		{
			shared_max[i] = 0;
		}

		for (int i = 0; i < rules.size(); ++i)
		{
			string l_sym = rules[i]->right1;
			string r_sym = rules[i]->right2;
			string symbol = rules[i]->left;

			int local_max = 0;
			int bpSplit=-1;
			string right1 = "";
			string right2 = "";

			for (int split =start+1;split<=end-1;split++){
				int lscore = scores[start][split][symIndices[l_sym]];
				int rscore = scores[split][end][symIndices[r_sym]];
				int score = rules[i]->score + lscore + rscore;

				if(score > local_max){
					local_max = score;

					printf("symbol : %d local_max : %d l_score : %d r_score : %d score : %d\n", symIndices[symbol], local_max, lscore, rscore, score);
					//needed for backpointer
					bpSplit = split;
					right1 = l_sym;
					right2 = r_sym;

				}
			}	

			//atomic max // for now use lock
			guard.lock();
			shared_max[symIndices[symbol]] = max(shared_max[symIndices[symbol]], local_max);
			bp[start][end][symIndices[symbol]]->setBP(right1,right2,bpSplit);
			guard.unlock();								
		}

	  	for (int i = 0; i < symbols.size(); ++i)
	  	{
	  		cout << shared_max[i] << " ";
	  	}
	  	cout << "\n@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n";


		//make this parallel for
		for (std::vector<string>::iterator itr = symbols.begin(); itr != symbols.end(); ++itr){
			scores[start][end][symIndices[*itr]] = max(scores[start][end][symIndices[*itr]], shared_max[symIndices[*itr]]);			
		}

			// scores[start][end][symIndices[*itr]]=local_max;
			// // cout << right1 << " " << right2 << endl;
			// bp[start][end][symIndices[*itr]]->setBP(right1,right2,bpSplit);

		unaryRelax(scores, start, end, rules,symbols, bp);
	}	
}




//Parallel Binary Relax
void blockBasedRuleBR(int *** scores, int nWords, int length, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp){

	for(int start =0; start <=nWords-length; start++){
		
		int end = start + length;
		int* shared_max = new int[symbols.size()];
		for (int i = 0; i < symbols.size(); ++i)
		{
			shared_max[i] = 0;
		}

		for (int i = 0; i < nWords; ++i)
		{
			for (int j = 0; j < nWords; ++j)
			{
				for (int k = 0; k < symbols.size(); ++k)
				{
					cout << scores[i][j][k] << " ";
				}
			}
		}

		cout << endl;
		cout << "\n******************************************\n";
	
		// #pragma cilk grainsize = 1
		//for all symbols
		//make the for parallel
		for (std::vector<string>::iterator itr = symbols.begin(); itr != symbols.end(); ++itr){
			int local_max = 0;
			RuleVector rulesList = rulesMap[*itr];
			string right1="", right2="";
			int bpSplit=-1;

			//For each binary rule in the grammar
			for(int j=0;j<rulesList.size();j++){

				//TODO check if for unary rules, the right child is in right1
				if(rulesList[j]->is_second_order()){
					for (int split =start+1;split<=end-1;split++){
						int lscore = scores[start][split][symIndices[rulesList[j]->right1]];
						int rscore = scores[split][end][symIndices[rulesList[j]->right2]];
						int score = rulesList[j]->score + lscore + rscore;

						if(score > local_max){
							local_max = score;

							//needed for backpointer
							bpSplit = split;
							right1 = rulesList[j]->right1;
							right2 = rulesList[j]->right2;

						}
					}
				}
			}

			//atomic max // for now use lock
			guard.lock();
			shared_max[symIndices[*itr]] = max(shared_max[symIndices[*itr]], local_max);
			bp[start][end][symIndices[*itr]]->setBP(right1,right2,bpSplit);
			guard.unlock();
		}

	  	for (int i = 0; i < symbols.size(); ++i)
	  	{
	  		cout << shared_max[i] << " ";
	  	}
	  	cout << "\n@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n";

		//make this parallel for
		for (std::vector<string>::iterator itr = symbols.begin(); itr != symbols.end(); ++itr){
			scores[start][end][symIndices[*itr]] = max(scores[start][end][symIndices[*itr]], shared_max[symIndices[*itr]]);	
		}




			// scores[start][end][symIndices[*itr]]=local_max;
			// // cout << right1 << " " << right2 << endl;
			// bp[start][end][symIndices[*itr]]->setBP(right1,right2,bpSplit);

		unaryRelax(scores, start, end, rules,symbols, bp);
	}
}




void lexiconScores(int*** scores, const StringVector & words, RuleVector & rules, RuleVector & lexicons, SymbolsSet& symbols, backpointer**** bp){
	cout<<endl<<endl<<endl<<"In Lexicon Scores"<<endl;
	RuleVector rulesList;
	std::vector<string> :: iterator itr;

	//for all words
	for(int i=0;i<words.size();i++){
		//for all symbols
		for (itr = symbols.begin(); itr != symbols.end(); ++itr){
			rulesList = lexiconsMap[*itr];
			//for all rules of that symbol
			for(int k=0;k<rulesList.size();k++){
				//if the unary rule produces that symbol on the RHS
				if(rulesList[k]->is_first_order()){
					if(rulesList[k]->right1 == words[i]){
						
						//add score of that rule in the location
						scores[i][i+1][symIndices[*itr]] = rulesList[k]->score;
						cout<<rulesList[k]->left<<" ------> "<<rulesList[k]->right1;
					}
				}
			}
				
		}
		cout<<endl;
		unaryRelax(scores, i, i+1, rules,symbols, bp);
		
	}
	cout<<"Out of lexiconScores";
}

void printMatrix(int *** scores, int x, int y, int z){
	for (int i = 0; i < x; ++i)
	{
		for (int j = 0; j < y; ++j)
		{
			for (int k = 0; k < z; ++k)
			{
				cout<<" "<< scores[i][j][k]<<" ";
			}
			cout<<endl;
		}
	}
}

void printBPTree(backpointer**** bp, int start, int end, string symbol, const StringVector & words){

	if (symIndices.find(symbol) == symIndices.end())
	{
		cout << symbol << "not found\n";
		return;
	}

	int symIndex = symIndices[symbol];
	backpointer* curr = bp[start][end][symIndex];

	cout << "%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%\n";

	cout << symbol << " -> " << curr->right1 <<" "<< curr->right2 << endl;
	if(curr->right1 =="" && curr->right2 ==""){
		cout<<words[start]<<endl;
	}

	cout << "%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%\n";

	int split = (curr->split != 0) ? curr->split : 0;

	if(curr->right1 != ""){
		printBPTree(bp, start, split, curr->right1,words);
	}

	if (curr->right2 != "")
	{
		printBPTree(bp, split, end, curr->right2,words);
	}

}

void fillMap(RulesMap& map, RuleVector& rules){
	string key;
	for(auto i = 0; i < rules.size(); ++i){
                key = rules[i]->left;
                if(map.find(key)  == map.end()){ // key not exist
                        map[key] = RuleVector {};
                }
                map[key].push_back(rules[i]);
        }
}	

#include <algorithm>
void fillSet(SymbolsSet& sym, RuleVector& rules){
	string key;
	for(auto i = 0; i < rules.size(); ++i){
            key = rules[i]->left;
            if (std::find(sym.begin(), sym.end(), key) == sym.end() )
            {
            	sym.push_back(key);
            }
    }
}	

void initializeRulesMap(RulesMap & map, RuleVector & rules, RuleVector & lexicons){
	
	fillMap(map,rules);
	//fillMap(map,lexicons);	
}

void initializeLexiconsMap(RulesMap & map, RuleVector & lexicons){
	fillMap(map,lexicons);	
}

void initializeSymbols(SymbolsSet& sym,RuleVector & rules, RuleVector & lexicons){
	fillSet(sym,rules);
	fillSet(sym,lexicons);
}

void initializeSymbolIndices(SymbolsSet& symbols, SymbolIndices& symIndices){

	int count =0;
	vector<string> :: iterator itr;
	for (itr = symbols.begin(); itr != symbols.end(); ++itr)
    {
    	symIndices[*itr] = count;
    	count++;
    }

}

struct SymbolMap{
	// string Symbol;
	int symNum;
	IntRule* i;
	int curEl;
};


SymbolMap* convertSymbolsMap(SymbolsSet& symbols)
{

	SymbolMap* symMap = new SymbolMap[symbols.size()];

	for (int i = 0; i < symbols.size(); ++i)
	{
		symMap[i].symNum = symIndices[symbols[i]];
		RuleVector rules = rulesMap[symbols[i]];
		for (int j = 0; j < rules.size(); ++j)
		{
			symMap[i].i = new IntRule[rules.size()];

			symMap[i].i[j].leftsymIndex = symIndices[rules[j]->left];
			symMap[i].i[j].right1symIndex = symIndices[rules[j]->right1];
			symMap[i].i[j].right2symIndex = symIndices[rules[j]->right2];
			symMap[i].i[j].score = rules[i]->score;
		}

	}
	return symMap;
}

SymbolMap* convertSymbolsMapToCudaDevice(SymbolMap* sm, int size){
  
  SymbolMap* dsm;
  hipMalloc((void**)&dsm, size*sizeof(SymbolMap));
  hipMemcpy(dsm, sm, size*sizeof(SymbolMap), hipMemcpyHostToDevice);	

  return dsm;
}

__global__
static void blockBasedBR(SymbolMap* dsm, int numSymbols, IntRule* drules, int numRules, int* dscores, int A, int B, int C, int start, int end, int* sh_max) {
  int i = blockIdx.x;
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  SymbolMap sm = dsm[i];
  IntRule* rulesList = sm.i;
  int lscore, rscore, score;
  int max = -1;
  int bpSplit;

  if(j<numRules){
	// if(rulesList[j].is_second_order()){
		for (int split =start+1;split<=end-1;split++){

			int l_sym = rulesList[j].right1symIndex;
			int r_sym = rulesList[j].right2symIndex;
			int symbol = rulesList[j].leftsymIndex;

			lscore = dscores[start + B * (split + C * l_sym)];
			rscore = dscores[split + B * (end + C * r_sym)];
			score = rulesList[j].score + lscore + rscore;

			if(score > max){
				max = score;
				bpSplit = split;
				}
			}
		// }
		atomicMax(&sh_max[sm.symNum], score);
	}	
}

void blockBasedBRUtil(int *** scores, int nWords, int length, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp, int A, int B, int C){

	cout<<"In block based BR";
	IntRule* hrules = ConvertRule(rules);
	IntRule* drules = ConvertRuleTOCudaDevice(hrules, rules.size());
	int* newScores;
	int end;

	int* shared_max = new int[symbols.size()];
		for (int i = 0; i < symbols.size(); ++i)
		{
			shared_max[i] = 0;
		}
	
	for(int start =0; start <=nWords-length; start++){
		
		end = start + length;
		//convert3DArrayToCudaDevice((int*)&scores, newScores,A,B,C);

		int* arr = ConvertScore(scores, A, B, C);
		int* dscores = ConvertToCudaDevice(arr, A*B*C);

		int* sh_max = ConvertToCudaDevice(shared_max, symbols.size());
		
		SymbolMap* sm = convertSymbolsMap(symbols);
		SymbolMap* dsm = convertSymbolsMapToCudaDevice(sm,symbols.size());
		cout<<"Entering Kernel"<<endl;
		blockBasedBR<<<symbols.size(), rules.size()*2>>>(dsm, symbols.size(), drules, rules.size(), dscores, A, B, C, start, end, sh_max);
		cout<<"Exiting Kernel"<<endl;
		hipMemcpy(shared_max, sh_max, symbols.size()*sizeof(int), hipMemcpyDeviceToHost);
	}

}

#include <unistd.h>
void cykParser(const StringVector & words, RuleVector & rules, RuleVector & lexicons, SymbolsSet& symbols){
	//size should be words.length, words.length, total non-terminals 

	cout<<" Size of words "<<words.size()+1<<endl;
	cout<<" Size of symbols "<<symbols.size()+1<<endl;

	int *** scores  = new int**[words.size()+1];
	backpointer**** bp = new backpointer***[words.size()+1];

	for (int i = 0; i < words.size()+1; ++i)
	{
		scores[i] = new int*[words.size()+1];
		bp[i] = new backpointer**[words.size()+1];

		for (int j = 0; j < words.size()+1; ++j)
		{
			scores[i][j] = new int[symbols.size()];
			bp[i][j] = new backpointer*[symbols.size()];


			for (int k = 0; k < symbols.size(); ++k)
			{
				scores[i][j][k]=0;
				bp[i][j][k]= new backpointer("","",-1);
			}
		}
	}

	//[words.size()+1][symbols.size()];
	int nWords = words.size();
		
	lexiconScores(scores,words,rules,lexicons,symbols,bp);

	cout<<"Out of LExicon Scores"<<endl;
	for(int length =2; length<=nWords; length++){
		// binaryRelax(scores,nWords,length,rules, symbols, bp);
		//blockBasedRuleBR(scores,nWords,length,rules, symbols, bp);
		// threadBasedRuleBRcpu(scores,nWords,length,rules, symbols, bp);
		cout<<"Word size"<<words.size()<< endl;
		cout<<"Symbols size"<<symbols.size()<< endl;
		cout<<"Calling Block BR "<<endl;
		blockBasedBRUtil(scores,nWords,length,rules, symbols, bp, words.size(), words.size(), symbols.size());
		//threadBasedRuleBR(scores,nWords,length,rules, symbols, bp, words.size(), words.size(), symbols.size());
	}

	printMatrix(scores, words.size()+1,words.size()+1,symbols.size());
	// printBPTree(bp, 0, words.size(), "S",words);
}

void cykParserUtil(const StringVector & words, RuleVector & rules, RuleVector & lexicons){
	
	//Write initialization code
	initializeRulesMap(rulesMap, rules, lexicons);
	initializeLexiconsMap(lexiconsMap,lexicons);
	initializeSymbols(symbols,rules,lexicons);
	initializeSymbolIndices(symbols, symIndices);
	cykParser(words,rules,lexicons,symbols);

}
#include "hip/hip_runtime.h"
#include "cyk.h"
#include <map>
#include <set>
// #include <cilk/cilk.h>
using namespace std;

typedef std::vector<string> SymbolsSet;
typedef std::map<string, RuleVector> RulesMap;
typedef std::map<string, int> SymbolIndices;
typedef std::map<string, RuleVector> LexiconsMap;

RulesMap rulesMap;
LexiconsMap lexiconsMap;
SymbolsSet symbols;
SymbolIndices symIndices;


void unaryRelax (int *** scores, int begin, int end, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp){
	cout<<endl<<endl<<endl<<"In Unary Relax"<<endl;
	int prob=0;
	RuleVector rulesList;
	std::vector<string>::iterator itr;
	
	//for all symbols
	for (itr = symbols.begin(); itr != symbols.end(); ++itr){
		rulesList = rulesMap[*itr];
		//for all rules of that symbol
		for(int k=0;k<rulesList.size();k++){
			//get only unary rules
			if(rulesList[k]->is_first_order()){

				//The rule is of the form A-> B. get the score of A>B and that symbol B is present at score [begin][end]
				prob = rulesList[k]->score + scores[begin][end][symIndices[rulesList[k]->right1]];

				// if the above score is greater than the score of B, then add A to the location [begin][end]
		     	if(prob > scores[begin][end][symIndices[rulesList[k]->left]]){
		     		scores[begin][end][symIndices[rulesList[k]->left]] = prob;
		     		// cout << rulesList[k]->left << " ----> "<<rulesList[k]->right1 << endl;
		     		bp[begin][end][symIndices[rulesList[k]->left]]->setBP(rulesList[k]->right1,"",0);
		     	}
         	}
		}
	}
}


void binaryRelax(int *** scores, int nWords, int length, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp){
	cout<<endl<<endl<<endl<<"In Binary Relax"<<endl;
	int end=0;
	int max = -1;
	RuleVector rulesList;
	int lscore=0, rscore =0,score=0;
	std::vector<string> :: iterator itr;
	string right1="", right2="";
	int bpSplit=-1;

	for(int start =0; start <=nWords-length; start++){
		
		end = start + length;
		
		//for all symbols
		for (itr = symbols.begin(); itr != symbols.end(); ++itr){
			max = 0;
			rulesList = rulesMap[*itr];
			for(int j=0;j<rulesList.size();j++){

				//TODO check if for unary rules, the right child is in right1
				if(rulesList[j]->is_second_order()){
					for (int split =start+1;split<=end-1;split++){
						lscore = scores[start][split][symIndices[rulesList[j]->right1]];
						rscore = scores[split][end][symIndices[rulesList[j]->right2]];
						score = rulesList[j]->score + lscore + rscore;

						if(score > max){
							max = score;
							bpSplit = split;
							right1 = rulesList[j]->right1;
							right2 = rulesList[j]->right2;
							// cout<<rulesList[j]->left<<" ------> "<<rulesList[j]->right1<<" "<<rulesList[j]->right2;
						}
					}
				}
			}
			scores[start][end][symIndices[*itr]]=max;
			bp[start][end][symIndices[*itr]]->setBP(right1,right2,bpSplit);

			// cout << right1 << " " << right2 << endl;
			
		}

		unaryRelax(scores, start, end, rules,symbols, bp);
	}
}

struct IntRule{
	int leftsymIndex;
	int right1symIndex;
	int right2symIndex;
	int score;
};


__global__
static void MyFunc2D(IntRule* drules, int r, int* dscores, int A, int B, int C, int start, int end, int* sh_max) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  	
  //Debug Code
  if (i == 0)
  {
	// for (int i = 0; i < r; ++i)
	// {
	// 	printf("%d -> %d %d\n", drules[i].leftsymIndex, drules[i].right1symIndex, drules[i].right2symIndex);
	// }
	// for (int u = 0; u < A; ++u)
	// {
	// 	for (int v = 0; v < B; ++v)
	// 	{
	// 	  for (int w = 0; w < C; ++w)
	// 	  {
	// 	    printf("%d ", dscores[u + B * (v + C * w)]);
	// 	  }
	// 	  printf("\n");
	// 	}
	// 	printf("\n");
	// }

  }



  if (i < r)
  {
  		  // printf("Index %d ", i);
		{
			int l_sym = drules[i].right1symIndex;
			int r_sym = drules[i].right2symIndex;
			int symbol = drules[i].leftsymIndex;

			int local_max = 0;
			// int bpSplit=-1;
			// string right1 = "";
			// string right2 = "";

			for (int split =start+1;split<=end-1;split++){
				int lscore = 0;
				int rscore = 0;

				// start + B * (split + C * l_sym)
				// https://stackoverflow.com/questions/7367770/how-to-flatten-or-index-3d-array-in-1d-array
				// (z * xMax * yMax) + (y * xMax) + x;
				int one = (l_sym*A*B) + (split*A) + start;	
				lscore = dscores[one];
	
				int two = (r_sym*A*B) + (end*A) + split;
				rscore = dscores[two];

				int score = drules[i].score + lscore + rscore;

				if(score > local_max){
					local_max = score;
					printf("symbol : %d local_max : %d l_score : %d r_score : %d score : %d\n", symbol, local_max, lscore, rscore, score);

					//needed for backpointer
					// bpSplit = split;
					// right1 = l_sym;
					// right2 = r_sym;

				}
			}	

			//atomic max // for now use lock

			atomicMax(&sh_max[symbol], local_max);
			// bp[start][end][symIndices[symbol]]->setBP(right1,right2,bpSplit);
		}		
  }
}



IntRule* ConvertRule(RuleVector& rules)
{
	IntRule* ruleArr = new IntRule[rules.size()];
	for (int i = 0; i < rules.size(); ++i)
	{
		ruleArr[i].leftsymIndex = symIndices[rules[i]->left];
		ruleArr[i].right1symIndex = symIndices[rules[i]->right1];
		ruleArr[i].right2symIndex = symIndices[rules[i]->right2];
		ruleArr[i].score = rules[i]->score;
	}

	return ruleArr;
}

int to1D( int x, int y, int z , int xMax, int yMax, int zMax) {
    return (z * xMax * yMax) + (y * xMax) + x;
}

int* ConvertScore(int*** score, int A, int B, int C)
{
  int* hscore;// = new int[A*B*C*10];
  hipHostMalloc((void**)&hscore, A*B*C*sizeof(int));


  for (int i = 0; i < A; ++i)
  {
    for (int j = 0; j < B; ++j)
    {
      for (int k = 0; k < C; ++k)
      {
        // hscore[i + B * (j + C * k)] = score[i][j][k];
      	hscore[to1D(i, j, k, A, B, C)] = score[i][j][k];
      }
    }
  }


  return hscore;
}

int* ConvertToCudaDevice(int* a, int n){
  int* d;
  hipMalloc((void**)&d, n*sizeof(int));
  hipMemcpy(d, a, n*sizeof(int), hipMemcpyHostToDevice);

  return d;
}

IntRule* ConvertRuleTOCudaDevice(IntRule* rules, int N){

  IntRule* drules;
  hipMalloc((void**)&drules, N*sizeof(IntRule));
  hipMemcpy(drules, rules, N*sizeof(IntRule), hipMemcpyHostToDevice);	

  return drules;
}

void threadBasedRuleBR(int *** scores, int nWords, int length, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp, int A, int B, int C){

	cout << "Baby i am here " <<  A << " " << B << " " << C << " " << symbols.size() << " " << nWords << endl; 

	IntRule* hrules = ConvertRule(rules);
	IntRule* drules = ConvertRuleTOCudaDevice(hrules, rules.size());


	for(int start =0; start <=nWords-length; start++){
		// for (int i = 0; i < A; ++i)
		// {
		// 	for (int j = 0; j < B; ++j)
		// 	{
		// 		for (int k = 0; k < C; ++k)
		// 		{
		// 			cout << scores[i][j][k] << " ";
		// 		}
		// 	}
		// }

		// cout << endl;
		// cout << "\n******************************************\n";


		int* arr = ConvertScore(scores, A, B, C);
		int* dscores = ConvertToCudaDevice(arr, A*B*C);
		// int* dscores;
		// hipMalloc((void**)&dscores, A*B*C*sizeof(int));
		// hipMemset(dscores, 0, A*B*C*sizeof(int));
		// hipMemcpy(dscores, arr, A*B*C*sizeof(int), hipMemcpyHostToDevice);

		
		int end = start + length;
		int* shared_max = new int[symbols.size()];
		for (int i = 0; i < symbols.size(); ++i)
		{
			shared_max[i] = 0;
		}

		int* sh_max = ConvertToCudaDevice(shared_max, symbols.size());

		// int** testy = alloc2d();
	  	MyFunc2D<<<1, rules.size()*2>>>(drules, rules.size(), dscores, A, B, C, start, end, sh_max);


	  	hipMemcpy(shared_max, sh_max, symbols.size()*sizeof(int), hipMemcpyDeviceToHost);

	  	for (int i = 0; i < symbols.size(); ++i)
	  	{
	  		cout << shared_max[i] << " ";
	  	}
	  	cout << "\n@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n";

		//make this parallel for
		for (std::vector<string>::iterator itr = symbols.begin(); itr != symbols.end(); ++itr){
			scores[start][end][symIndices[*itr]] = max(scores[start][end][symIndices[*itr]], shared_max[symIndices[*itr]]);	
		}



		unaryRelax(scores, start, end, rules,symbols, bp);
	}	
}


#include <mutex>
std::mutex guard;
void threadBasedRuleBRcpu(int *** scores, int nWords, int length, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp){




	for(int start =0; start <=nWords-length; start++){
		
		int end = start + length;
		int* shared_max = new int[symbols.size()];
		for (int i = 0; i < symbols.size(); ++i)
		{
			shared_max[i] = 0;
		}

		for (int i = 0; i < rules.size(); ++i)
		{
			string l_sym = rules[i]->right1;
			string r_sym = rules[i]->right2;
			string symbol = rules[i]->left;

			int local_max = 0;
			int bpSplit=-1;
			string right1 = "";
			string right2 = "";

			for (int split =start+1;split<=end-1;split++){
				int lscore = scores[start][split][symIndices[l_sym]];
				int rscore = scores[split][end][symIndices[r_sym]];
				int score = rules[i]->score + lscore + rscore;

				if(score > local_max){
					local_max = score;

					printf("symbol : %d local_max : %d l_score : %d r_score : %d score : %d\n", symIndices[symbol], local_max, lscore, rscore, score);
					//needed for backpointer
					bpSplit = split;
					right1 = l_sym;
					right2 = r_sym;

				}
			}	

			//atomic max // for now use lock
			guard.lock();
			shared_max[symIndices[symbol]] = max(shared_max[symIndices[symbol]], local_max);
			bp[start][end][symIndices[symbol]]->setBP(right1,right2,bpSplit);
			guard.unlock();								
		}

	  	for (int i = 0; i < symbols.size(); ++i)
	  	{
	  		cout << shared_max[i] << " ";
	  	}
	  	cout << "\n@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n";


		//make this parallel for
		for (std::vector<string>::iterator itr = symbols.begin(); itr != symbols.end(); ++itr){
			scores[start][end][symIndices[*itr]] = max(scores[start][end][symIndices[*itr]], shared_max[symIndices[*itr]]);			
		}

			// scores[start][end][symIndices[*itr]]=local_max;
			// // cout << right1 << " " << right2 << endl;
			// bp[start][end][symIndices[*itr]]->setBP(right1,right2,bpSplit);

		unaryRelax(scores, start, end, rules,symbols, bp);
	}	
}




//Parallel Binary Relax
void blockBasedRuleBR(int *** scores, int nWords, int length, RuleVector& rules, SymbolsSet& symbols, backpointer**** bp){

	for(int start =0; start <=nWords-length; start++){
		
		int end = start + length;
		int* shared_max = new int[symbols.size()];
		for (int i = 0; i < symbols.size(); ++i)
		{
			shared_max[i] = 0;
		}

		for (int i = 0; i < nWords; ++i)
		{
			for (int j = 0; j < nWords; ++j)
			{
				for (int k = 0; k < symbols.size(); ++k)
				{
					cout << scores[i][j][k] << " ";
				}
			}
		}

		cout << endl;
		cout << "\n******************************************\n";
	
		// #pragma cilk grainsize = 1
		//for all symbols
		//make the for parallel
		for (std::vector<string>::iterator itr = symbols.begin(); itr != symbols.end(); ++itr){
			int local_max = 0;
			RuleVector rulesList = rulesMap[*itr];
			string right1="", right2="";
			int bpSplit=-1;

			//For each binary rule in the grammar
			for(int j=0;j<rulesList.size();j++){

				//TODO check if for unary rules, the right child is in right1
				if(rulesList[j]->is_second_order()){
					for (int split =start+1;split<=end-1;split++){
						int lscore = scores[start][split][symIndices[rulesList[j]->right1]];
						int rscore = scores[split][end][symIndices[rulesList[j]->right2]];
						int score = rulesList[j]->score + lscore + rscore;

						if(score > local_max){
							local_max = score;

							//needed for backpointer
							bpSplit = split;
							right1 = rulesList[j]->right1;
							right2 = rulesList[j]->right2;

						}
					}
				}
			}

			//atomic max // for now use lock
			guard.lock();
			shared_max[symIndices[*itr]] = max(shared_max[symIndices[*itr]], local_max);
			bp[start][end][symIndices[*itr]]->setBP(right1,right2,bpSplit);
			guard.unlock();
		}

	  	for (int i = 0; i < symbols.size(); ++i)
	  	{
	  		cout << shared_max[i] << " ";
	  	}
	  	cout << "\n@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n";

		//make this parallel for
		for (std::vector<string>::iterator itr = symbols.begin(); itr != symbols.end(); ++itr){
			scores[start][end][symIndices[*itr]] = max(scores[start][end][symIndices[*itr]], shared_max[symIndices[*itr]]);	
		}




			// scores[start][end][symIndices[*itr]]=local_max;
			// // cout << right1 << " " << right2 << endl;
			// bp[start][end][symIndices[*itr]]->setBP(right1,right2,bpSplit);

		unaryRelax(scores, start, end, rules,symbols, bp);
	}
}




void lexiconScores(int*** scores, const StringVector & words, RuleVector & rules, RuleVector & lexicons, SymbolsSet& symbols, backpointer**** bp){
	cout<<endl<<endl<<endl<<"In Lexicon Scores"<<endl;
	RuleVector rulesList;
	std::vector<string> :: iterator itr;

	//for all words
	for(int i=0;i<words.size();i++){
		//for all symbols
		for (itr = symbols.begin(); itr != symbols.end(); ++itr){
			rulesList = lexiconsMap[*itr];
			//for all rules of that symbol
			for(int k=0;k<rulesList.size();k++){
				//if the unary rule produces that symbol on the RHS
				if(rulesList[k]->is_first_order()){
					if(rulesList[k]->right1 == words[i]){
						
						//add score of that rule in the location
						scores[i][i+1][symIndices[*itr]] = rulesList[k]->score;
						cout<<rulesList[k]->left<<" ------> "<<rulesList[k]->right1;
					}
				}
			}
				
		}
		cout<<endl;
		unaryRelax(scores, i, i+1, rules,symbols, bp);
		
	}
}

void printMatrix(int *** scores, int x, int y, int z){
	for (int i = 0; i < x; ++i)
	{
		for (int j = 0; j < y; ++j)
		{
			for (int k = 0; k < z; ++k)
			{
				cout<<" "<< scores[i][j][k]<<" ";
			}
			cout<<endl;
		}
	}
}

void printBPTree(backpointer**** bp, int start, int end, string symbol, const StringVector & words){

	if (symIndices.find(symbol) == symIndices.end())
	{
		cout << symbol << "not found\n";
		return;
	}

	int symIndex = symIndices[symbol];
	backpointer* curr = bp[start][end][symIndex];

	cout << "%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%\n";

	cout << symbol << " -> " << curr->right1 <<" "<< curr->right2 << endl;
	if(curr->right1 =="" && curr->right2 ==""){
		cout<<words[start]<<endl;
	}

	cout << "%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%\n";

	int split = (curr->split != 0) ? curr->split : 0;

	if(curr->right1 != ""){
		printBPTree(bp, start, split, curr->right1,words);
	}

	if (curr->right2 != "")
	{
		printBPTree(bp, split, end, curr->right2,words);
	}

}
#include <unistd.h>
void cykParser(const StringVector & words, RuleVector & rules, RuleVector & lexicons, SymbolsSet& symbols){
	//size should be words.length, words.length, total non-terminals 

	cout<<" Size of words "<<words.size()+1<<endl;
	cout<<" Size of symbols "<<symbols.size()+1<<endl;

	int *** scores  = new int**[words.size()+1];
	backpointer**** bp = new backpointer***[words.size()+1];

	for (int i = 0; i < words.size()+1; ++i)
	{
		scores[i] = new int*[words.size()+1];
		bp[i] = new backpointer**[words.size()+1];

		for (int j = 0; j < words.size()+1; ++j)
		{
			scores[i][j] = new int[symbols.size()];
			bp[i][j] = new backpointer*[symbols.size()];


			for (int k = 0; k < symbols.size(); ++k)
			{
				scores[i][j][k]=0;
				bp[i][j][k]= new backpointer("","",-1);
			}
		}
	}

	//[words.size()+1][symbols.size()];
	int nWords = words.size();
		
	lexiconScores(scores,words,rules,lexicons,symbols,bp);

	for(int length =2; length<=nWords; length++){
		// binaryRelax(scores,nWords,length,rules, symbols, bp);
		// blockBasedRuleBR(scores,nWords,length,rules, symbols, bp);
		// threadBasedRuleBRcpu(scores,nWords,length,rules, symbols, bp);

		threadBasedRuleBR(scores,nWords,length,rules, symbols, bp, words.size()+1, words.size()+1, symbols.size());
	}

	printMatrix(scores, words.size()+1,words.size()+1,symbols.size());
	// printBPTree(bp, 0, words.size(), "S",words);
}

void fillMap(RulesMap& map, RuleVector& rules){
	string key;
	for(auto i = 0; i < rules.size(); ++i){
                key = rules[i]->left;
                if(map.find(key)  == map.end()){ // key not exist
                        map[key] = RuleVector {};
                }
                map[key].push_back(rules[i]);
        }
}	

#include <algorithm>
void fillSet(SymbolsSet& sym, RuleVector& rules){
	string key;
	for(auto i = 0; i < rules.size(); ++i){
            key = rules[i]->left;
            if (std::find(sym.begin(), sym.end(), key) == sym.end() )
            {
            	sym.push_back(key);
            }
    }
}	

void initializeRulesMap(RulesMap & map, RuleVector & rules, RuleVector & lexicons){
	
	fillMap(map,rules);
	//fillMap(map,lexicons);	
}

void initializeLexiconsMap(RulesMap & map, RuleVector & lexicons){
	fillMap(map,lexicons);	
}

void initializeSymbols(SymbolsSet& sym,RuleVector & rules, RuleVector & lexicons){
	fillSet(sym,rules);
	fillSet(sym,lexicons);
}

void initializeSymbolIndices(SymbolsSet& symbols, SymbolIndices& symIndices){

	int count =0;
	vector<string> :: iterator itr;
	for (itr = symbols.begin(); itr != symbols.end(); ++itr)
    {
    	symIndices[*itr] = count;
    	count++;
    }

}

void cykParserUtil(const StringVector & words, RuleVector & rules, RuleVector & lexicons){
	
	//Write initialization code
	initializeRulesMap(rulesMap, rules, lexicons);
	initializeLexiconsMap(lexiconsMap,lexicons);
	initializeSymbols(symbols,rules,lexicons);
	initializeSymbolIndices(symbols, symIndices);
	cykParser(words,rules,lexicons,symbols);

}